#include <ctype.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <sys/times.h>
#include <time.h>
#include <hip/hip_runtime.h>
#define PI 3.14159265358979323846
#define FactorArcosegRad 0.00000484814

clock_t timestart, timeend;

#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600
#else
__device__ double atomicAdd(double* a, double b) { return b; }
#endif

/**
@brief Función que transforma un valor en arco segundo a radianes
@param deltax: Valor numérico a transformar
@returns Valor correspondiente a la entrada en radianes */
double arcoseg_radian(double deltax){
	return FactorArcosegRad*deltax;
}

/**
@brief Función que lee el archivo de entrada
@param archivo: puntero al archivo a leer
@param archivo: puntero al archivo a leer
@returns  */
double* readFile(FILE* archivo, int tamano){
	double* elementos =(double*) malloc(sizeof(double)*4*tamano);
	fread(elementos, tamano*4, sizeof(double), archivo);
	return elementos;
}

__global__ 
void gridding_process(double *X, double *Y, double *R, double *I, int num_datos, int tamano, double deltaU, double *r, double *k){

	long i;
	double x, y, modx, mody;
	for (i = threadIdx.x; i < num_datos; i+=blockDim.x)
	{
		x = X[i]/deltaU+tamano/2;
		y = Y[i]/deltaU+tamano/2;
		modx = X[i] - x*deltaU;
		mody = Y[i] - y*deltaU;
		if(modx>deltaU/2){	
			x+=1;
		}
		if (mody>deltaU/2)
		{
			y+=1;
		}
		//r[(int)y*tamano+(int)x] += R[i];
		//k[(int)y*tamano+(int)x] += I[i];
		atomicAdd(&r[(int)y*tamano+(int)x], R[i]);
		atomicAdd(&k[(int)y*tamano+(int)x], I[i]);
	}
}

int main(int argc, char * const argv[])
{
	int tamano;//tamaño de imagen
	int numdatos;//número de pasos
	double deltaX_arcoseg, deltaX_radian;
	double deltaU; 
	char* archivo_entrada=NULL;
	char* archivo_salida=NULL;
	int i, c;
	opterr = 0;
	while ((c = getopt (argc, argv, "i:z:d:N:o:")) != -1)
		switch (c)
			{
			case 'i':
				archivo_entrada = optarg;
				break;
			case 'z':
				numdatos = atoi(optarg);
				break;
			case 'd':
				deltaX_arcoseg = atof(optarg);
				break;
			case 'N':
				tamano = atoi(optarg);
				break;
			case 'o':
				archivo_salida = optarg;
				break;
			case '?':
				if (optopt == 'i' ||optopt == 'z' ||optopt == 'd'||optopt == 'N' ||optopt == 'o')
					fprintf (stderr, "Opcion -%c requiere un argumento.\n", optopt);
				else if (isprint (optopt))
					fprintf (stderr, "Opcion desconocida `-%c'.\n", optopt);
				else
					fprintf (stderr,
									"Carater opcion desconocido `\\x%x'.\n",
									optopt);
				return 1;
			default:
				abort ();
			}
	/**
		Comprobación de Inputs
			- Valores mayores que cero
			- Cadenas no nulas
	**/
	if(tamano<=0){
		printf("El parametro -N debe estár y ser mayor que 0\n");
		exit(1);
	}
	if(numdatos==0){
		printf("El parametro -z debe estár y ser mayor que 0\n");
		exit(1);
	}
	if(deltaX_arcoseg==0){
		printf("El parametro -d debe estár y ser mayor que 0\n");
		exit(1);
	}
	if(archivo_entrada==NULL){
		printf("Debe especificarse un archivo de entrada\n");
	}
	if(archivo_salida==NULL){
		printf("Debe especificarse un archivo de salida\n");
	}
	//Transformacion de unidades necesaria para calcular delta U
	deltaX_radian = arcoseg_radian(deltaX_arcoseg);

	//Determina delta U/V a utilizar
	deltaU = 1/(tamano*deltaX_radian);

	//Medición de tiempo de computo
	timestart = clock(); 

	//Lectura de entrada
	FILE *entrada = fopen(archivo_entrada,"r");
	double* data = readFile(entrada,numdatos);
	fclose(entrada);

	//Creando arrays para coordenada X, Y, R e I
	double *X = (double*)malloc(sizeof(double)*numdatos); 
	double *Y = (double*)malloc(sizeof(double)*numdatos); 
	double *R = (double*)malloc(sizeof(double)*numdatos); 
	double *I = (double*)malloc(sizeof(double)*numdatos);	
	//Quizas necesite dos vectores adicionales para el gridding [matrices desenroyadas]
	double *r = (double*)malloc(sizeof(double)*tamano*tamano);
	double *k = (double*)malloc(sizeof(double)*tamano*tamano);
	//Se asigan los valores correspondientes de la lectura
	for (i = 0; i < numdatos; i++)
	{
		X[i] = data[i];
		Y[i] = data[i+numdatos];
		R[i] = data[i+2*numdatos];
		I[i] = data[i+3*numdatos];

	}
	for (i = 0; i < tamano*tamano; ++i)
	{
		r[i] = 0;
		k[i] = 0;
	}
	//se declaran las variables CUDA
	double *C_X;
	double *C_Y;
	double *C_R;
	double *C_I;
	double *C_r;
	double *C_k;
	//Se reserva memoria CUDA
	hipMalloc( (void**)&C_X, numdatos*sizeof(double)); 
	hipMalloc( (void**)&C_Y, numdatos*sizeof(double)); 
	hipMalloc( (void**)&C_R, numdatos*sizeof(double)); 
	hipMalloc( (void**)&C_I, numdatos*sizeof(double)); 
	hipMalloc( (void**)&C_r, tamano*tamano*sizeof(double)); 
	hipMalloc( (void**)&C_k, tamano*tamano*sizeof(double)); 
	//se copia la matriz iniciada en las matrices de trabajo en memoria global GPU
	hipMemcpy( C_X, X, numdatos*sizeof(double), hipMemcpyHostToDevice); 
	hipMemcpy( C_Y, Y, numdatos*sizeof(double), hipMemcpyHostToDevice); 
	hipMemcpy( C_R, R, numdatos*sizeof(double), hipMemcpyHostToDevice); 
	hipMemcpy( C_I, I, numdatos*sizeof(double), hipMemcpyHostToDevice); 
	//Se declaran las dimenciones
	dim3 dimBlock(1, 32);
	dim3 dimGrid(1, 1);
	//se ejecuta el kernel en la GPU
	gridding_process<<<dimGrid, dimBlock>>>(C_X, C_Y, C_R, C_Y, numdatos, tamano, deltaU, C_r, C_k);
	//se espera a que terminen
	hipDeviceSynchronize();
	//se obtiene la memoria de regreso
	hipMemcpy( r, C_r, tamano*tamano*sizeof(double), hipMemcpyDeviceToHost); 
	hipMemcpy( k, C_k, tamano*tamano*sizeof(double), hipMemcpyDeviceToHost); 
	//se libera la memoria global CUDA para que pueda ser usada por otro proceso
	hipFree( C_X );
	hipFree( C_Y );
	hipFree( C_R );
	hipFree( C_I );
	hipFree( C_r );
	hipFree( C_k );
	//Se imprime salida
	FILE *f = fopen("salida_real","wb");
	FILE *g = fopen("salida_imaginaria","wb");

	fwrite(r,tamano*tamano, sizeof(double),f);
	fwrite(k,tamano*tamano, sizeof(double),g);

	timeend = clock(); // registramos el tiempo hasta el final
	printf("Total = %f\n", (double) (timeend-timestart)/(double)CLOCKS_PER_SEC);
	return EXIT_SUCCESS;
}