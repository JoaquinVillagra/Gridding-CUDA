#include <ctype.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <sys/times.h>
#include <time.h>
#include <hip/hip_runtime.h>
#define PI 3.14159265358979323846
#define FactorArcosegRad 0.00000484814

clock_t timestart, timeend;

/**
@brief Función que transforma un valor en arco segundo a radianes
@param deltax: Valor numérico a transformar
@returns Valor correspondiente a la entrada en radianes */
double arcoseg_radian(double deltax){
	return FactorArcosegRad*deltax;
}

/**
@brief Función que lee el archivo de entrada
@param archivo: puntero al archivo a leer
@param archivo: puntero al archivo a leer
@returns  */
double* readFile(FILE* archivo, int tamano){
	double* elementos =(double*) malloc(sizeof(double)*4*tamano);
	fread(elementos, tamano*4, sizeof(double), archivo);
	return elementos;
}

__global__ 
void gridding_process(double *X, double *Y, int num_datos, int tamano, double deltaU, int *G){

	__shared__ double x_s[32], y_s[32], g_s[32];
	long i, pos;
	double x, y, modx, mody;
	if(threadIdx.x==0)
	{
		for (i = 0; i < 32; i++)
		{
			x_s[i] = X[i+blockIdx.x*32];
			y_s[i] = Y[i+blockIdx.x*32];
		}
	}
	pos = blockIdx.x*32 + threadIdx.x;
	
	x = x_s[threadIdx.x]/deltaU+tamano/2;
	y = y_s[threadIdx.x]/deltaU+tamano/2;
	modx = x_s[threadIdx.x] - x*deltaU;
	mody = y_s[threadIdx.x] - y*deltaU;
	if(modx>deltaU/2){	
		x+=1;
	}
	if (mody>deltaU/2)
	{
		y+=1;
	}

	g_s[pos] = (int)y*tamano+(int)x;

	__syncthreads();
	if (threadIdx.x==0)
	{
		for (i = 0; i < 32; i++)
		{
			G[i+blockIdx.x*32] = g_s[i];
		}
	}
}

int main(int argc, char * const argv[])
{
	int tamano;//tamaño de imagen
	int numdatos;//número de pasos
	double deltaX_arcoseg, deltaX_radian;
	double deltaU; 
	char* archivo_entrada=NULL;
	char* archivo_salida=NULL;
	int i, c;
	opterr = 0;
	while ((c = getopt (argc, argv, "i:z:d:N:o:")) != -1)
		switch (c)
			{
			case 'i':
				archivo_entrada = optarg;
				break;
			case 'z':
				numdatos = atoi(optarg);
				break;
			case 'd':
				deltaX_arcoseg = atof(optarg);
				break;
			case 'N':
				tamano = atoi(optarg);
				break;
			case 'o':
				archivo_salida = optarg;
				break;
			case '?':
				if (optopt == 'i' ||optopt == 'z' ||optopt == 'd'||optopt == 'N' ||optopt == 'o')
					fprintf (stderr, "Opcion -%c requiere un argumento.\n", optopt);
				else if (isprint (optopt))
					fprintf (stderr, "Opcion desconocida `-%c'.\n", optopt);
				else
					fprintf (stderr,
									"Carater opcion desconocido `\\x%x'.\n",
									optopt);
				return 1;
			default:
				abort ();
			}
	/**
		Comprobación de Inputs
			- Valores mayores que cero
			- Cadenas no nulas
	**/
	if(tamano<=0){
		printf("El parametro -N debe estár y ser mayor que 0\n");
		exit(1);
	}
	if(numdatos==0){
		printf("El parametro -z debe estár y ser mayor que 0\n");
		exit(1);
	}
	if(deltaX_arcoseg==0){
		printf("El parametro -d debe estár y ser mayor que 0\n");
		exit(1);
	}
	if(archivo_entrada==NULL){
		printf("Debe especificarse un archivo de entrada\n");
	}
	if(archivo_salida==NULL){
		printf("Debe especificarse un archivo de salida\n");
	}
	//Transformacion de unidades necesaria para calcular delta U
	deltaX_radian = arcoseg_radian(deltaX_arcoseg);

	//Determina delta U/V a utilizar
	deltaU = 1/(tamano*deltaX_radian);

	//Medición de tiempo de computo
	timestart = clock(); 

	//Lectura de entrada
	FILE *entrada = fopen(archivo_entrada,"r");
	double* data = readFile(entrada,numdatos);
	fclose(entrada);

	//Creando arrays para coordenada X, Y, R e I
	double *X = (double*)malloc(sizeof(double)*numdatos); 
	double *Y = (double*)malloc(sizeof(double)*numdatos); 
	double *R = (double*)malloc(sizeof(double)*numdatos); 
	double *I = (double*)malloc(sizeof(double)*numdatos);
	int *G = (int*)malloc(sizeof(int)*numdatos);	
	//Quizas necesite dos vectores adicionales para el gridding [matrices desenroyadas]
	double *r = (double*)malloc(sizeof(double)*tamano*tamano);
	double *k = (double*)malloc(sizeof(double)*tamano*tamano);
	//Se asigan los valores correspondientes de la lectura
	for (i = 0; i < numdatos; i++)
	{
		X[i] = data[i];
		Y[i] = data[i+numdatos];
		R[i] = data[i+2*numdatos];
		I[i] = data[i+3*numdatos];
		G[i] = 0;
	}
	for (i = 0; i < tamano*tamano; ++i)
	{
		r[i] = 0;
		k[i] = 0;
	}
	//se declaran las variables CUDA
	double *C_X;
	double *C_Y;
	double *C_R;
	double *C_I;
	int *C_G;
	//Se reserva memoria CUDA
	hipMalloc( (void**)&C_X, numdatos*sizeof(double)); 
	hipMalloc( (void**)&C_Y, numdatos*sizeof(double)); 
	hipMalloc( (void**)&C_R, numdatos*sizeof(double)); 
	hipMalloc( (void**)&C_I, numdatos*sizeof(double)); 
	hipMalloc( (void**)&C_G, numdatos*sizeof(int));
	//se copia la matriz iniciada en las matrices de trabajo en memoria global GPU
	hipMemcpy( C_X, X, numdatos*sizeof(double), hipMemcpyHostToDevice); 
	hipMemcpy( C_Y, Y, numdatos*sizeof(double), hipMemcpyHostToDevice); 
	hipMemcpy( C_R, R, numdatos*sizeof(double), hipMemcpyHostToDevice); 
	hipMemcpy( C_I, I, numdatos*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy( C_G, G, numdatos*sizeof(int), hipMemcpyHostToDevice); 
	//Se declaran las dimenciones
	dim3 dimBlock(32, 1);
	dim3 dimGrid(1, 1);
	//se ejecuta el kernel en la GPU
	gridding_process<<<dimGrid, dimBlock>>>(C_X, C_Y, numdatos, tamano, deltaU, C_G);
	//se espera a que terminen
	hipDeviceSynchronize();
	//se obtiene la memoria de regreso
	hipMemcpy( G, C_G, numdatos*sizeof(double), hipMemcpyDeviceToHost); 
	//se libera la memoria global CUDA para que pueda ser usada por otro proceso
	hipFree( C_X );
	hipFree( C_Y );
	hipFree( C_R );
	hipFree( C_I );
	hipFree( C_G );
	//Secuencialmente se hace reducción a la posicion
	for (i = 0; i < numdatos; i++)
	{
		printf("G = %d\n", G[i] );
		r[G[i]] += R[i];
		k[G[i]] += I[i];
	}

	//Se imprime salida
	FILE *f = fopen("salida_real","wb");
	FILE *g = fopen("salida_imaginaria","wb");

	fwrite(r,tamano*tamano, sizeof(double),f);
	fwrite(k,tamano*tamano, sizeof(double),g);

	timeend = clock(); // registramos el tiempo hasta el final
	printf("Total = %f\n", (double) (timeend-timestart)/(double)CLOCKS_PER_SEC);
	return EXIT_SUCCESS;
}